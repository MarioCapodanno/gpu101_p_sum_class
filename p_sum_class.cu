#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h> // stampare tempo esecuzione
#include <stdlib.h>   // malloc

#define DIM 204800

// Check API call
#define CHECK(call)                                            \
  {                                                            \
    const hipError_t err = call;                              \
    if (err != hipSuccess)                                    \
    {                                                          \
      printf("%s in %s at line %d\n", hipGetErrorString(err), \
             __FILE__, __LINE__);                              \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }

// Check to kernel call
#define CHECK_KERNELCALL()                                     \
  .                                                            \
  {                                                            \
    const hipError_t err = hipGetLastError();                \
    if (err != hipSuccess)                                    \
    {                                                          \
      printf("%s in %s at line %d\n", hipGetErrorString(err), \
             __FILE__, __LINE__);                              \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }

double get_time()
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}

void p_sum_cpu(float *p_sum, float *input, int length)
{
  p_sum[0] = input[0];
  int i;
  for (i = 1; i < length; ++i)
  {
    p_sum[i] = p_sum[i - 1] + input[i - 1];
  }
}

__global__ void p_sum_gpu(float *p_sum, float *input, int length)
{
  p_sum[0] = input[0];
  int i;
  for (i = 1; i < length; ++i)
  {
    p_sum[i] = p_sum[i - 1] + input[i - 1];
  }
}

int main(int argc, char *argv[])
{

  int i;
  double start_cpu, end_cpu, start_gpu, end_gpu;

  srand(time(NULL));

  float *p_sum_sw = (float *)malloc(sizeof(float) * DIM);
  float *input_v = (float *)malloc(sizeof(float) * DIM);
  float *p_sum_hw = (float *)malloc(sizeof(float) * DIM);

  for (i = 0; i < DIM; i++)
  {
    input_v[i] = rand() % 100; // genera numeri da 0 a 99
  }

  float *input_d, *p_sum_d;

  CHECK(hipMalloc(&input_d, sizeof(float) * DIM));
  CHECK(hipMalloc(&p_sum_d, sizeof(float) * DIM));

  start_cpu = get_time();
  p_sum_cpu(p_sum_sw, input_v, DIM);
  end_cpu = get_time();

  CHECK(hipMemcpy(input_d, input_v, sizeof(float) * DIM, hipMemcpyHostToDevice));

  start_gpu = get_time();
  dim3 blockPerGrid(1, 1, 1);
  dim3 threadsPerBlock(1, 1, 1);
  p_sum_gpu<<<blockPerGrid, threadsPerBlock>>>(p_sum_d, input_d, DIM);
  CHECK_KERNELCALL();
  CHECK(hipDeviceSynchronize());
  end_gpu = get_time();

  CHECK(hipMemcpy(p_sum_hw, p_sum_d, sizeof(float) * DIM, hipMemcpyDeviceToHost));

  if (p_sum_hw[DIM - 1] != p_sum_sw[DIM - 1])
  {
    fprintf(stderr, "ERRORE RISULTATO SBAGLIATO SU GPU1\n");
  }

  printf("GPU TIME: %lf, CPU TIME: %lf", end_gpu - start_gpu, end_cpu - start_cpu);
  CHECK(hipFree(input_d));
  CHECK(hipFree(p_sum_d));

  free(p_sum_sw);
  free(input_v);
  free(p_sum_hw);

  return 0;
}